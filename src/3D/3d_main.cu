#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <hipfft/hipfft.h>
#include <vector>
#include <algorithm>

#include "check_correct.hpp"
#include "helper_cuda/hip/hip_runtime_api.h"

#define n_unit 1
#define unit (8 * n_unit)
#define rfft_size (unit * unit * unit)

#define nwarp_in_block 1

#include "rfft_3d/3d_rfft_N.cu"

#define bank_unit (unit + 2)
#define bank_unit_unit (bank_unit * bank_unit )
#define shared_size (unit * bank_unit_unit)
// #include "rfft_3d/3d_rfft_N_bank.cu"


void printHelp()
{
    const char *helpMessage =
        "Program name: FlashFFTStencil-3D\n"
        "Usage: a.out [stencil-shape] [input_size] [time_step] \n"
        "Stencil-shape: Heat-3D \n";
    printf("%s\n", helpMessage);
}


int main(int argc, char *argv[])
{
    if (argc != 4)
    {
        printHelp();
        return 1;
    }

    const int INPUT_WIDTH = 768;
    const int KERNEL_WIDTH = 3;
    std::string kernel_shape = argv[1];
    const int time = std::stoi(argv[3]);
    const bool is_print_data = false;

    const int sub_input_width = unit - (KERNEL_WIDTH - 1); // TODO : 6, 14, 22, 30
    // const int sub_input_width = unit;
    const int OVERLAP_WIDTH = KERNEL_WIDTH - 1;

    if (INPUT_WIDTH % sub_input_width != 0)
    {
        std::cerr << "input_size % subinput_size != 0" << std::endl;
        std::cerr << "subinput_size = " << sub_input_width << std::endl;
        std::cerr << "input_size = " << INPUT_WIDTH << std::endl;
        return 0.0;
    }
    // else
    // {
    //     std::cout << "INFO: stencil kernel shape = " << kernel_shape << std::endl;
    //     std::cout << "INFO: input width = " << INPUT_WIDTH << std::endl;
    //     std::cout << "INFO: times step = " << time << std::endl;
    // }

    const unsigned int block_num_x = (INPUT_WIDTH / sub_input_width);
    const unsigned int block_num_y = (INPUT_WIDTH / sub_input_width);
    const unsigned int block_num_z = (INPUT_WIDTH / sub_input_width) / 2;

    const int ACTUAL_WIDTH = (INPUT_WIDTH / sub_input_width) * unit;

    const int gpu_input_size = (INPUT_WIDTH / sub_input_width) * (INPUT_WIDTH / sub_input_width) * (INPUT_WIDTH / sub_input_width) * rfft_size;
    const int cpu_input_size = (INPUT_WIDTH * INPUT_WIDTH * INPUT_WIDTH);
    const int kernel_size = KERNEL_WIDTH * KERNEL_WIDTH * KERNEL_WIDTH;

    // malloc
    size_t mem_size_input_gpu = gpu_input_size * sizeof(double);
    size_t mem_size_output = cpu_input_size * sizeof(double);

    double *h_input_gpu = (double *)calloc(gpu_input_size, sizeof(double));

    double *h_input_cpu = (double *)calloc(cpu_input_size, sizeof(double));

    double *h_output = (double *)calloc(cpu_input_size, sizeof(double));

    std::vector<double> h_kernel(kernel_size);

    // 初始化输入数据
    for (int i = 0; i < INPUT_WIDTH; i++)
    {
        for (int j = 0; j < INPUT_WIDTH; j++)
        {
            for (int k = 0; k < INPUT_WIDTH; k++)
            {
                if (is_print_data)
                {
                    h_input_cpu[i * INPUT_WIDTH * INPUT_WIDTH + j * INPUT_WIDTH + k] = static_cast<double>(1);
                }
                else
                {
                    h_input_cpu[i * INPUT_WIDTH * INPUT_WIDTH + j * INPUT_WIDTH + k] = static_cast<double>(rand()) / static_cast<double>(RAND_MAX);
                }

                int index_for_inputgpu = ((i / sub_input_width) * unit + i % sub_input_width) * ACTUAL_WIDTH * ACTUAL_WIDTH + ((j / sub_input_width) * unit + j % sub_input_width) * ACTUAL_WIDTH + ((k / sub_input_width) * unit + k % sub_input_width);
                h_input_gpu[index_for_inputgpu] = h_input_cpu[i * INPUT_WIDTH * INPUT_WIDTH + j * INPUT_WIDTH + k];
            }
        }
    }
    for (int i = 0; i < KERNEL_WIDTH; i++)
    {
        for (int j = 0; j < KERNEL_WIDTH; j++)
        {
            for (int k = 0; k < KERNEL_WIDTH; k++)
            {
                if (is_print_data)
                {
                    h_kernel[i * KERNEL_WIDTH * KERNEL_WIDTH + j * KERNEL_WIDTH + k] = static_cast<double>((1));
                }
                else
                {
                    h_kernel[i * KERNEL_WIDTH * KERNEL_WIDTH + j * KERNEL_WIDTH + k] = static_cast<double>(rand()) / static_cast<double>(RAND_MAX);
                }
            }
        }
    }

    CreatePlan(h_kernel.data(), KERNEL_WIDTH, is_print_data);

    // malloc device memory
    double *d_input;
    checkCudaErrors(hipMalloc((void **)&d_input, mem_size_input_gpu));
    double *d_output;
    checkCudaErrors(hipMalloc((void **)&d_output, mem_size_output));
    checkCudaErrors(hipMemset(d_output, 0, mem_size_output));

    checkCudaErrors(hipMemcpy(d_input, h_input_gpu, mem_size_input_gpu, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start, 0));

    for (int i= 0; i < time; i++)
    {


    rfft_3d_8_nwarp<nwarp_in_block><<<
        {block_num_x, block_num_y, block_num_z},
        // {1, 1, 1},

        nwarp_in_block * WARP_SIZE * n_unit,

        (nwarp_in_block * 2 * shared_size) * sizeof(double)
        // (nwarp_in_block * 2 * rfft_size) * sizeof(double)
        
        >>>(

        d_input,
        ACTUAL_WIDTH,
        INPUT_WIDTH,
        sub_input_width,
        OVERLAP_WIDTH,
        // fft_allnum - 1,
        d_output);

    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    // compute time
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    // std::cout << "Time = " << elapsedTime << "[ms]" << std::endl;

    std::string kernel_shape_output;
    if (kernel_shape == "Heat-3D")
        kernel_shape_output = "star_3d1r";
    else if (kernel_shape == "Box3D27P")
        kernel_shape_output = "box_3d1r";

    std::cout << "FlashFFTStencil, " << kernel_shape_output << ", " << INPUT_WIDTH << ", " << INPUT_WIDTH << ", " << INPUT_WIDTH << ", " << time << ", " << elapsedTime << ", " << ((double)INPUT_WIDTH * INPUT_WIDTH * INPUT_WIDTH * time) / elapsedTime / 1e6 << std::endl;

    hipMemcpy(h_output, d_output, mem_size_output, hipMemcpyDeviceToHost);

    free(h_input_cpu);
    free(h_input_gpu);
    free(h_output);

    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}