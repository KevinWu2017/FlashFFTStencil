#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <hipfft/hipfft.h>
#include <vector>
#include <algorithm>

#include "check_correct.hpp"
#include "helper_cuda/hip/hip_runtime_api.h"

#define unit 8
#define rfft_size (unit * unit)

#define band_unit (8 )
#define shared_unit (unit * band_unit)

#define nwarp_in_block 1

// #include "rfft_2d/2d_rfft_1_naive.cu"
// #include "rfft_2d/2d_rfft_1_banksolve.cu"

#include "rfft_2d/2d_rfft_1_async.cu"

// #include "rfft_2d/2d_rfft_1_naive_2warp.cu"

// #include "rfft_2d/2d_rfft_2.cu"
// #include "rfft_2d/2d_rfft_2_banksolve.cu"

void printHelp()
{
    const char *helpMessage =
        "Program name: FlashFFTStencil-2D\n"
        "Usage: a.out [stencil-shape] [input_size] [time_step] \n"
        "Stencil-shape: Heat-2D / Box2D9P \n";
    printf("%s\n", helpMessage);
}


int main(int argc, char *argv[])
{

    if (argc != 4)
    {
        printHelp();
        return 1;
    }

    std::string kernel_shape = argv[1];
    const int INPUT_WIDTH = std::stoi(argv[2]);
    const int KERNEL_WIDTH = 3;
    const int time = std::stoi(argv[3]);

    const bool is_print_data = false;

    const int sub_input_width = unit - (KERNEL_WIDTH - 1); // TODO : 54,
    // const int sub_input_width = unit;
    const int OVERLAP_WIDTH = KERNEL_WIDTH - 1;

    if (INPUT_WIDTH % sub_input_width != 0)
    {
        std::cerr << "input_size % subinput_size != 0" << std::endl;
        std::cerr << "subinput_size = " << sub_input_width << std::endl;
        std::cerr << "input_size = " << INPUT_WIDTH << std::endl;
        return 0.0;
    }
    // else
    // {
    //     std::cout << "INFO: stencil kernel shape = " << kernel_shape << std::endl;
    //     std::cout << "INFO: input width = " << INPUT_WIDTH << std::endl;
    //     std::cout << "INFO: times step = " << time << std::endl;
    // }

    const unsigned int block_num_x = (INPUT_WIDTH / sub_input_width) / 2 / nwarp_in_block;
    const unsigned int block_num_y = (INPUT_WIDTH / sub_input_width);

    const int ACTUAL_WIDTH = (INPUT_WIDTH / sub_input_width) * unit;

    const int gpu_input_size = (INPUT_WIDTH / sub_input_width) * (INPUT_WIDTH / sub_input_width) * rfft_size;
    const int cpu_input_size = (INPUT_WIDTH * INPUT_WIDTH);
    const int kernel_size = KERNEL_WIDTH * KERNEL_WIDTH;

    // malloc
    size_t mem_size_input_gpu = gpu_input_size * sizeof(double);
    size_t mem_size_output = cpu_input_size * sizeof(double);

    double *h_input_gpu = (double *)calloc(gpu_input_size, sizeof(double));

    double *h_input_cpu = (double *)calloc(cpu_input_size, sizeof(double));

    double *h_output = (double *)calloc(cpu_input_size, sizeof(double));

    std::vector<double> h_kernel(kernel_size);

    // 初始化输入数据
    for (int i = 0; i < INPUT_WIDTH; i++)
    {
        for (int j = 0; j < INPUT_WIDTH; j++)
        {
            if (is_print_data)
            {
                h_input_cpu[i * INPUT_WIDTH + j] = static_cast<double>(1);
            }
            else
            {
                h_input_cpu[i * INPUT_WIDTH + j] = static_cast<double>(rand()) / static_cast<double>(RAND_MAX);
            }

            int index_for_inputgpu = ((i / sub_input_width) * unit + i % sub_input_width) * ACTUAL_WIDTH + ((j / sub_input_width) * unit + j % sub_input_width);
            h_input_gpu[index_for_inputgpu] = h_input_cpu[i * INPUT_WIDTH + j];
        }
    }
    for (int i = 0; i < KERNEL_WIDTH; i++)
    {
        for (int j = 0; j < KERNEL_WIDTH; j++)
        {
            if (is_print_data)
            {
                h_kernel[i * KERNEL_WIDTH + j] = static_cast<double>((1));
            }
            else
            {
                h_kernel[i * KERNEL_WIDTH + j] = static_cast<double>(rand()) / static_cast<double>(RAND_MAX);
            }
        }
    }

    CreatePlan(h_kernel.data(), KERNEL_WIDTH, is_print_data);

    // malloc device memory
    double *d_input;
    checkCudaErrors(hipMalloc((void **)&d_input, mem_size_input_gpu));
    double *d_output;
    checkCudaErrors(hipMalloc((void **)&d_output, mem_size_output));
    checkCudaErrors(hipMemcpy(d_input, h_input_gpu, mem_size_input_gpu, hipMemcpyHostToDevice));

    // checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(rfft_2d_8_nwarp<nwarp_in_block>), hipFuncCachePreferL1)); // 最差
    // checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(rfft_2d_8_nwarp<nwarp_in_block>), hipFuncCachePreferShared));
    // checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(rfft_2d_8_nwarp<nwarp_in_block>), hipFuncCachePreferEqual));

    // checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(rfft_2d_8_nwarp<nwarp_in_block>), hipFuncAttributeMaxDynamicSharedMemorySize, (nwarp_in_block * 2 * shared_unit) * sizeof(double)));

    int time_i = 0;
    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));

    for(; time_i < time; time_i++)
    {
    rfft_2d_8_nwarp<nwarp_in_block><<<
        {block_num_x, block_num_y},
        // 1,
        nwarp_in_block * WARP_SIZE,

        (nwarp_in_block * 2 * shared_unit) * sizeof(double)

        // (nwarp_in_block * 2 * rfft_size) * sizeof(double)

        >>>(

        d_input,
        ACTUAL_WIDTH,
        INPUT_WIDTH,
        sub_input_width,
        OVERLAP_WIDTH,
        // fft_allnum - 1,
        d_output);
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    // compute time
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    // std::cout << "Time = " << elapsedTime << "[ms]" << std::endl;

    std::string kernel_shape_output;
    if (kernel_shape == "Heat-2D")
        kernel_shape_output = "star_2d1r";
    else if (kernel_shape == "Box2D9P")
        kernel_shape_output = "box_2d1r";

    std::cout << "FlashFFTStencil, " << kernel_shape_output << ", 1, " << INPUT_WIDTH << ", " << INPUT_WIDTH << ", " << time << ", " << elapsedTime << ", " << ((double)INPUT_WIDTH * INPUT_WIDTH * time) / elapsedTime / 1e6 << std::endl;

    hipMemcpy(h_output, d_output, mem_size_output, hipMemcpyDeviceToHost);

    // // 计算CPU结果
    // std::vector<std::vector<double>> new_kernel = rotateKernel180(h_kernel);
    // std::vector<std::vector<double>> new_input = convertTo2D(h_input_cpu, INPUT_WIDTH);
    // std::vector<double> ref_result = stencil2D(new_input, new_kernel);

    // if (is_print_data)
    // {
    //     std::ofstream outfile("output.txt");

    //     if (!outfile)
    //     {
    //         std::cerr << "无法打开文件!" << std::endl;
    //         return -1;
    //     }

    //     outfile << "输入数据：" << std::endl;
    //     for (int i = 0; i < ACTUAL_WIDTH; i++)
    //     {
    //         for (int j = 0; j < ACTUAL_WIDTH; j++)
    //         {
    //             outfile << h_input_gpu[i * ACTUAL_WIDTH + j] << ", ";
    //         }
    //         outfile << std::endl;
    //     }
    //     outfile << std::endl;

    //     outfile << "Kernel: " << std::endl;
    //     for (int i = 0; i < KERNEL_WIDTH; i++)
    //     {
    //         for (int j = 0; j < KERNEL_WIDTH; j++)
    //         {
    //             outfile << h_kernel[i * KERNEL_WIDTH + j] << "\t";
    //         }
    //         outfile << std::endl;
    //     }
    //     outfile << std::endl;

    //     outfile << "GPU计算结果：" << std::endl;
    //     for (int i = 0; i < INPUT_WIDTH; i++)
    //     {
    //         for (int j = 0; j < INPUT_WIDTH; j++)
    //         {
    //             outfile << h_output[i * INPUT_WIDTH + j] << "\t";
    //         }
    //         outfile << std::endl;
    //     }
    //     outfile << std::endl;

    //     outfile << "new input: " << std::endl;
    //     for (int i = 0; i < INPUT_WIDTH; i++)
    //     {
    //         for (int j = 0; j < INPUT_WIDTH; j++)
    //         {
    //             outfile << new_input[i][j] << "\t";
    //         }
    //         outfile << std::endl;
    //     }

    //     outfile << "new kernel: " << std::endl;
    //     for (int i = 0; i < KERNEL_WIDTH; i++)
    //     {
    //         for (int j = 0; j < KERNEL_WIDTH; j++)
    //         {
    //             outfile << new_kernel[i][j] << "\t";
    //         }
    //         outfile << std::endl;
    //     }

    //     outfile << "ref result: " << std::endl;
    //     for (int i = 0; i < INPUT_WIDTH; i++)
    //     {
    //         for (int j = 0; j < INPUT_WIDTH; j++)
    //         {
    //             outfile << ref_result[i * INPUT_WIDTH + j] << "\t";
    //         }
    //         outfile << std::endl;
    //     }

    //     outfile.close();
    // }

    // // 检查结果
    // if (areArraysEqual(h_output, ref_result.data(), INPUT_WIDTH * INPUT_WIDTH, 1e-6))
    // {
    //     std::cout << "Check correct!" << std::endl;
    // }
    // else
    // {
    //     std::cout << "Error: result wrong!" << std::endl;
    // }

    free(h_input_cpu);
    free(h_input_gpu);
    free(h_output);

    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}